#include <torch/extension.h>
#include <thrust/device_ptr.h>
#include <thrust/gather.h>
#include <thrust/scan.h>
#include <thrust/reduce.h>
#include <thrust/transform_reduce.h>
#include <thrust/transform_scan.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/iterator/discard_iterator.h>
#include "vb_matrices.h"
#include "thrust_allocator.h"

namespace cuda_playground {

template <typename index_t>
index_t get_total_size(
    index_t batch_size,
    thrust::device_ptr<index_t> m_ptr,
    thrust::device_ptr<index_t> n_ptr) {
  hipStream_t stream = at::cuda::getCurrentCUDAStream();
  ThrustAllocator allocator;
  auto policy = thrust::cuda::par(allocator).on(stream);
  
  return thrust::transform_reduce(
      policy,
      thrust::make_zip_iterator(thrust::make_tuple(m_ptr, n_ptr)),
      thrust::make_zip_iterator(thrust::make_tuple(m_ptr + batch_size, n_ptr + batch_size)),
      [] __device__ (thrust::tuple<index_t, index_t> tuple) {
        index_t m, n;
        thrust::tie(m, n) = tuple;
        return m * n;
      },
      0,
      thrust::plus<index_t>());
}

VBMatrices::VBMatrices(const std::vector<at::Tensor>& matrices) {
  batch_size_ = matrices.size();
  at::Tensor m_cpu = at::empty({batch_size_ + 1}, at::kInt);  // TODO: kInt -> index_t
  at::Tensor n_cpu = at::empty({batch_size_ + 1}, at::kInt);

  auto m_ptr = m_cpu.data_ptr<index_t>();
  auto n_ptr = n_cpu.data_ptr<index_t>();

  index_t data_size = 0;
  index_t last_n = -1;
  bool same_n = true;
  for (index_t i = 0; i < batch_size_; i++) {
    const auto& matrix = matrices[i];
    index_t m = matrix.size(0), n = matrix.size(1);
    m_ptr[i] = m;
    n_ptr[i] = n;
    data_size += m * n;

    if (last_n != -1 && last_n != n) {
      same_n = false;
    }
    last_n = n;
  }

  if (same_n) {
    data_ = at::empty({data_size / last_n, last_n}, matrices[0].options());
  } else {
    data_ = at::empty({data_size}, matrices[0].options());
  }

  
  m_cpu_ = m_cpu;
  n_cpu_ = n_cpu;
  m_ = m_cpu_.to(m_cpu_.options().device(data_.device()));
  n_ = n_cpu_.to(n_cpu_.options().device(data_.device()));

  hipStream_t stream = at::cuda::getCurrentCUDAStream();
  ThrustAllocator allocator;
  auto policy = thrust::cuda::par(allocator).on(stream);

  AT_DISPATCH_FLOATING_TYPES(matrices[0].scalar_type(), "VBMatrices::VBMatrices(const std::vector<at::Tensor> matrices)", [&] {
    auto data_ptr = data_.data_ptr<scalar_t>();

    for (index_t i = 0; i < batch_size_; i++) {
      const auto& matrix = matrices[i];
      index_t m = m_ptr[i], n = n_ptr[i];
      index_t size = m * n;
      auto data_i_ptr = matrix.data_ptr<scalar_t>();
      thrust::copy(policy, data_i_ptr, data_i_ptr + size, data_ptr);
      data_ptr += size;
    }
  });
}

void VBMatrices::reset(index_t batch_size, const at::Tensor &m, const at::Tensor &n, const at::TensorOptions &options) {
  reset(batch_size, batch_size, m, n, options);
}

void VBMatrices::reset(
    index_t batch_size,
    index_t num_groups,
    const at::Tensor &m,
    const at::Tensor &n,
    const at::TensorOptions &options,
    std::optional<at::Tensor> group_sizes,
    bool zero_init) {
  batch_size_ = batch_size;
  num_groups_ = num_groups;
  m_ = m;
  n_ = n;

  if (group_sizes.has_value()) {
    group_sizes_ = group_sizes.value();
  }

  auto m_ptr = thrust::device_ptr<index_t>(m.data_ptr<index_t>());
  auto n_ptr = thrust::device_ptr<index_t>(n.data_ptr<index_t>());

  auto data_size = get_total_size(batch_size, m_ptr, n_ptr);
  if (zero_init) {
    data_ = at::zeros({data_size}, options);
  } else {
    data_ = at::empty({data_size}, options);
  }

  // TODO: reset cached tensor
}

// work around the limitation of the lambda function in cuda
template <typename index_t>
void get_offsets_impl_thrust(
    const thrust::device_ptr<index_t> &m_ptr,
    const thrust::device_ptr<index_t> &n_ptr,
    thrust::device_ptr<index_t> &offsets_ptr,
    int batch_size) {
  hipStream_t stream = at::cuda::getCurrentCUDAStream();
  ThrustAllocator allocator;
  auto policy = thrust::cuda::par(allocator).on(stream);

  thrust::transform_exclusive_scan(
      policy,
      thrust::make_zip_iterator(thrust::make_tuple(m_ptr, n_ptr)),
      thrust::make_zip_iterator(thrust::make_tuple(m_ptr + batch_size, n_ptr + batch_size)),
      offsets_ptr,
      [] __device__ (const thrust::tuple<index_t, index_t> &tuple) {
        index_t m, n;
        thrust::tie(m, n) = tuple;
        return m * n;
      },
      0,
      thrust::plus<index_t>());
}

at::Tensor VBMatrices::get_offsets_impl() const {
  at::Tensor offsets = at::empty({batch_size_}, m_.options());

  auto m_ptr = thrust::device_ptr<index_t>(m_.data_ptr<index_t>());
  auto n_ptr = thrust::device_ptr<index_t>(n_.data_ptr<index_t>());
  auto offsets_ptr = thrust::device_ptr<index_t>(offsets.data_ptr<index_t>());

  get_offsets_impl_thrust<index_t>(m_ptr, n_ptr, offsets_ptr, batch_size_);

  return offsets;
}

// work around the limitation of the lambda function in cuda
template <typename scalar_t, typename index_t>
void get_addresses_impl_thrust(
    scalar_t *data_ptr,
    const thrust::device_ptr<index_t> &offsets_ptr,
    thrust::device_ptr<scalar_t *> &addresses_ptr,
    int batch_size) {
  hipStream_t stream = at::cuda::getCurrentCUDAStream();
  ThrustAllocator allocator;
  auto policy = thrust::cuda::par(allocator).on(stream);

  thrust::transform(
      policy,
      offsets_ptr,
      offsets_ptr + batch_size,
      addresses_ptr,
      [data_ptr] __device__ (index_t offset) {
        return data_ptr + offset;
      });
}

at::Tensor VBMatrices::get_addresses_impl() const {
  const auto& offsets = this->offsets();
  at::Tensor addresses = at::empty({batch_size_}, m_.options().dtype(at::kLong));

  AT_DISPATCH_FLOATING_TYPES(data_.scalar_type(), "VBMatrices::get_addresses_impl", [&] {
    auto data_ptr = data_.data_ptr<scalar_t>();
    auto offsets_ptr = thrust::device_ptr<index_t>(offsets.data_ptr<index_t>());
    auto addresses_ptr = thrust::device_ptr<scalar_t *>(reinterpret_cast<scalar_t **>(addresses.data_ptr<int64_t>()));

    get_addresses_impl_thrust(data_ptr, offsets_ptr, addresses_ptr, batch_size_);
  });

  return addresses;
}

template <typename index_t>
std::tuple<std::vector<index_t>, std::vector<index_t>> pack_up_impl(
    index_t batch_size,
    thrust::device_ptr<index_t> m_ptr,
    thrust::device_ptr<index_t> n_ptr,
    const at::TensorOptions& options) {
  hipStream_t stream = at::cuda::getCurrentCUDAStream();
  ThrustAllocator allocator;
  auto policy = thrust::cuda::par(allocator).on(stream);

  auto pack_sizes = at::empty(batch_size, options);
  auto packed_sizes = at::empty(batch_size, options);
  auto packed_m = at::empty(batch_size, options);
  auto reduced_packed_m = at::empty(batch_size, options);

  auto pack_sizes_ptr = thrust::device_ptr<index_t>(pack_sizes.template data_ptr<index_t>());
  auto packed_sizes_ptr = thrust::device_ptr<index_t>(packed_sizes.template data_ptr<index_t>());
  auto packed_m_ptr = thrust::device_ptr<index_t>(packed_m.template data_ptr<index_t>());
  auto reduced_packed_m_ptr = thrust::device_ptr<index_t>(reduced_packed_m.template data_ptr<index_t>());

  thrust::transform(policy, m_ptr, m_ptr + batch_size, packed_m_ptr, [] __device__ (index_t m) {
    if (m <= 64) {
      return 64;
    } else if (m <= 128) {
      return 128;
    } else if (m <= 256) {
      return 256;
    }
    return m;
  });

  auto new_end = thrust::reduce_by_key(
      policy,
      packed_m_ptr, packed_m_ptr + batch_size,
      thrust::make_constant_iterator<index_t>(1),
      thrust::make_discard_iterator(),
      pack_sizes_ptr);

  auto num_pack = thrust::distance(pack_sizes_ptr, new_end.second);
  std::cout << "num_pack: " << num_pack << std::endl << std::flush;

  std::vector<index_t> pack_sizes_cpu(num_pack), packed_sizes_cpu(num_pack);
  thrust::copy(pack_sizes_ptr, pack_sizes_ptr + num_pack, pack_sizes_cpu.begin());
  std::cout << "here #1" << std::endl << std::flush;
  thrust::copy(packed_sizes_ptr, packed_sizes_ptr + num_pack, packed_sizes_cpu.begin());
  std::cout << "here #2" << std::endl << std::flush;

  return {
    pack_sizes_cpu,
    packed_sizes_cpu,
  };
}

std::tuple<std::vector<VBMatrices::index_t>, std::vector<at::Tensor>> VBMatrices::pack_up() const {
  hipStream_t stream = at::cuda::getCurrentCUDAStream();
  ThrustAllocator allocator;
  auto policy = thrust::cuda::par(allocator).on(stream);
  
  auto options = m_.options();

  auto m = m_.clone();
  auto n = at::empty({batch_size_ + 1}, options);
  auto sorted_indices = at::arange(batch_size_, options);

  auto m_ptr = thrust::device_ptr<index_t>(m.data_ptr<index_t>());
  auto n_ptr = thrust::device_ptr<index_t>(n.data_ptr<index_t>());
  auto n__ptr = thrust::device_ptr<index_t>(n_.data_ptr<index_t>());
  auto sorted_indices_ptr = thrust::device_ptr<index_t>(sorted_indices.data_ptr<index_t>());

  thrust::sort_by_key(
      policy,
      m_ptr, m_ptr + batch_size_,
      sorted_indices_ptr);
  
  thrust::gather(
      policy,
      sorted_indices_ptr,
      sorted_indices_ptr + batch_size_,
      n__ptr,
      n_ptr);

  auto m_cpu = m.cpu();
  auto n_cpu = n.cpu();
  
  auto m_cpu_ptr = m_cpu.data_ptr<index_t>();
  auto n_cpu_ptr = n_cpu.data_ptr<index_t>();

  auto pack_sizes_and_packed_sizes = pack_up_impl<index_t>(batch_size_, m_ptr, n_ptr, options);
  auto pack_sizes = std::get<0>(pack_sizes_and_packed_sizes), packed_sizes = std::get<1>(pack_sizes_and_packed_sizes);
  std::vector<at::Tensor> packed_matrices;
  std::cout << "here #2.5" << std::endl << std::flush;

  AT_DISPATCH_FLOATING_TYPES(data_.scalar_type(), "VBMatrices::pack_up", [&] {
    std::cout << "here #2.6" << std::endl << std::flush;
    auto data_ptr = thrust::device_ptr<scalar_t>(data_.data_ptr<scalar_t>());
    const index_t num_pack = packed_sizes.size();
    index_t index_offset = 0;
    std::cout << "here #2.7" << std::endl << std::flush;

    for (index_t i = 0; i < num_pack; i++) {
      auto pack_size = pack_sizes[i], packed_size = packed_sizes[i];
      std::cout << "here #2.8" << std::endl << std::flush;
      auto m = m_cpu_ptr[index_offset];
      auto n = n_cpu_ptr[index_offset];
      auto size = pack_size * m * n;

      std::cout << "here #3" << std::endl << std::flush;
      auto matrix = at::empty({packed_size, m, n}, data_.options());
      auto matrix_ptr = thrust::device_ptr<scalar_t>(matrix.data_ptr<scalar_t>());
      thrust::copy(data_ptr, data_ptr + size, matrix_ptr);
      std::cout << "here #4" << std::endl << std::flush;
      packed_matrices.push_back(std::move(matrix));

      index_offset += pack_size;
      data_ptr += size;
    }
  });

  std::cout << "here done" << std::endl << std::flush;

  return {
    pack_sizes,
    packed_matrices
  };
}

namespace {
  template <typename policy_t, typename index_t>
  inline void generate_padded_m(
      const policy_t& policy,
      thrust::device_ptr<index_t> m_ptr,
      index_t batch_size,
      thrust::device_ptr<index_t> padded_m_ptr,
      const std::vector<index_t>& delimeters) {
    thrust::transform(policy, m_ptr, m_ptr + batch_size, padded_m_ptr, [=] __device__ (index_t m) {
      // for (auto delimeter : delimeters) {
      //   if (m <= delimeter) {
      //     return delimeter;
      //   }
      // }
      if (m <= 64) {
        return 64;
      } else if (m <= 128) {
        return 128;
      } else if (m <= 256) {
        return 256;
      }
      return m;
    });
  }

  template <typename policy_t, typename index_t>
  inline void generate_indices_and_masks(
      const policy_t& policy,
      index_t batch_size,
      thrust::device_ptr<index_t> unsorted_m_offsets_ptr,
      thrust::device_ptr<index_t> inverse_sorted_indices_ptr,
      thrust::device_ptr<index_t> padded_m_offsets_ptr,
      thrust::device_ptr<index_t> unsorted_m_ptr,
      thrust::device_ptr<index_t> indices_ptr,
      thrust::device_ptr<index_t> masks_ptr) {
    thrust::for_each(
        policy,
        thrust::make_counting_iterator<index_t>(0),
        thrust::make_counting_iterator<index_t>(batch_size),
        [=] __device__ (index_t i) {
          auto m_offset = unsorted_m_offsets_ptr[i];
          auto m = unsorted_m_ptr[i];
          auto idx = inverse_sorted_indices_ptr[i];
          auto padded_m_offset = padded_m_offsets_ptr[idx];
          thrust::sequence(
              policy,
              indices_ptr + m_offset, indices_ptr + m_offset + m,
              static_cast<index_t>(padded_m_offset));
          thrust::fill(policy, masks_ptr + m_offset, masks_ptr + m_offset + m, static_cast<index_t>(1));
        });
  }
}


std::tuple<VBMatrices, at::Tensor> VBMatrices::group_by() const {
  if (data_.dim() != 2) {
    throw std::runtime_error("VBMatrices::group_by() only supports 2D tensors");
  }

  auto policy = thrust::cuda::par(ThrustAllocator()).on(at::cuda::getCurrentCUDAStream());
  
  auto options = m_.options();

  auto m = m_.clone();
  auto m_ptr = thrust::device_ptr<index_t>(m.data_ptr<index_t>());
  
  auto sorted_indices = at::arange(batch_size_, options);
  auto sorted_indices_ptr = thrust::device_ptr<index_t>(sorted_indices.data_ptr<index_t>());
  
  thrust::sort_by_key(
      policy,
      m_ptr, m_ptr + batch_size_,
      sorted_indices_ptr);
  
  auto inverse_sorted_indices = at::empty_like(sorted_indices);
  auto inverse_sorted_indices_ptr = thrust::device_ptr<index_t>(inverse_sorted_indices.data_ptr<index_t>());
  thrust::gather(
      policy,
      sorted_indices_ptr, sorted_indices_ptr + batch_size_,
      thrust::make_counting_iterator<index_t>(0),
      inverse_sorted_indices_ptr);

  // TODO: do dp
  const std::vector<index_t> delimeters{64, 128, 256};

  auto padded_m = at::empty_like(m);
  auto padded_m_ptr = thrust::device_ptr<index_t>(padded_m.data_ptr<index_t>());
  generate_padded_m(policy, m_ptr, batch_size_, padded_m_ptr, delimeters);

  auto padded_m_offsets = at::empty_like(m);
  auto padded_m_offsets_ptr = thrust::device_ptr<index_t>(padded_m_offsets.data_ptr<index_t>());

  thrust::exclusive_scan(
      policy,
      padded_m_ptr, padded_m_ptr + batch_size_,
      padded_m_offsets_ptr);
  
  auto unsorted_m_ptr = thrust::device_ptr<index_t>(m_.data_ptr<index_t>());
  auto unsorted_m_offsets = at::empty_like(m);
  auto unsorted_m_offsets_ptr = thrust::device_ptr<index_t>(unsorted_m_offsets.data_ptr<index_t>());

  auto total_size = thrust::reduce(policy, m_ptr, m_ptr + batch_size_);
  auto indices = at::empty({total_size}, options);
  auto indices_ptr = thrust::device_ptr<index_t>(indices.data_ptr<index_t>());
  
  auto masks = at::empty({total_size}, options);
  auto masks_ptr = thrust::device_ptr<index_t>(masks.data_ptr<index_t>());

  generate_indices_and_masks(
      policy,
      batch_size_,
      unsorted_m_offsets_ptr,
      inverse_sorted_indices_ptr,
      padded_m_offsets_ptr,
      unsorted_m_ptr,
      indices_ptr,
      masks_ptr);

  auto group_m = at::empty({static_cast<int64_t>(delimeters.size())}, options);
  auto group_m_ptr = thrust::device_ptr<index_t>(group_m.data_ptr<index_t>());

  auto group_sizes = at::empty({static_cast<int64_t>(delimeters.size())}, options);
  auto group_sizes_ptr = thrust::device_ptr<index_t>(group_sizes.data_ptr<index_t>());

  auto new_end = thrust::reduce_by_key(
      policy,
      padded_m_ptr, padded_m_ptr + batch_size_,
      thrust::make_constant_iterator<index_t>(1),
      group_m_ptr,
      group_sizes_ptr);

  index_t num_groups = thrust::distance(group_sizes_ptr, new_end.second);

  VBMatrices grouped_matrices;

  auto group_m_cpu = group_m.cpu();
  auto group_n_cpu = at::full_like(group_m_cpu, data_.size(1));
  auto group_sizes_cpu = group_sizes.cpu();
  grouped_matrices.reset(batch_size_, num_groups, group_m_cpu, group_n_cpu, data_.options(), group_sizes_cpu, true);
  grouped_matrices.data().index_put_({indices, "..."}, data_);

  return {
    std::move(grouped_matrices),
    std::move(masks)
  };
}

} // namespace cuda_playground
